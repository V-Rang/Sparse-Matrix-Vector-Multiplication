/*
Naive mat-vec on CSR format matrix where each thread performs computations for a single row.
*/

#include<hip/hip_runtime.h>

#include<iostream>
#include<fstream>
#include<stdio.h>
#include<stdlib.h>
#include<string>
#include<algorithm>
#include<numeric>
#include<fstream>

using namespace std;

void sorter_result(int size_x, int *x, int *y, double *z, int *sorted_x, int *sorted_y, double *sorted_z)
{
    int *idx = new int[size_x];
    std::iota(idx,idx+size_x,0);

    stable_sort(idx, idx+size_x, [&x](int i1, int i2){ return x[i1] < x[i2]; });

    for(int i=0;i<size_x;i++)
    {
        sorted_x[i] = x[idx[i]];
        sorted_y[i] = y[idx[i]];
        sorted_z[i] = z[idx[i]];

    }
    delete idx;
    idx = nullptr;
}

__global__ void check()
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Hello from thread %d\n",tid);
}


// calc<<<nb,nt>>>(d_result,d_mat_vals,d_vec_vals,d_row_offset,d_col_inds,no_rows);
__global__ void calc(double *res, double *mat, double *vec, int *row_offset, int *col_inds, int no_rows)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(row < no_rows)
	{
		res[row] = 0;
		for(int i=row_offset[row]; i< row_offset[row+1]; i++)
		{
			res[row] += mat[i] * vec[col_inds[i]];
		}
	}

}
		

int main(int argc, char **argv)
{
	int num_times_to_run = 1;
	if(argc > 1)
	{
		num_times_to_run = atoi(argv[1]);
	}

	int no_rows, no_cols, nnz;
	int *row_ind, *col_ind;
	double *values;

	int*sorted_rows, *sorted_cols,*row_offset;
	double *sorted_vals;
	double *x_glob;

	string filename = "C:\\Users\\venu1\\OneDrive\\Desktop\\Work\\Sparse_Matrix_Vector_Multiplication\\Matrix_Files\\piston.mtx"; // any file in matrix market format, tested matrices taken from UF Sparse matrix collection.
	std::ifstream my_file;
	my_file.open(filename);
	string line;
	while(true)
	{
		if(my_file.eof()) break;
		getline(my_file,line);
		if(isdigit(line[0]) == 0) 
		{
			continue;
		}
		sscanf(line.c_str(),"%d %d %d",&no_rows,&no_cols,&nnz);
		break; 
	}
	
	row_ind = new int[nnz];
	col_ind = new int[nnz];
	values = new double[nnz];

	int index_keeper = 0;
	int row_ind_p1;
	int col_ind_p1;

	while(true)
	{
		if(my_file.eof()) break;
		getline(my_file,line);
		if(my_file.eof()) break; 
		sscanf(line.c_str(),"%d %d %lf",&row_ind_p1,&col_ind_p1,&values[index_keeper]);
		row_ind[index_keeper] = row_ind_p1-1;
		col_ind[index_keeper] = col_ind_p1-1;
		index_keeper += 1;
	}

	my_file.close();

	sorted_rows = new int[nnz];
	sorted_cols = new int[nnz];
	sorted_vals = new double[nnz];

	sorter_result(nnz,row_ind,col_ind,values,sorted_rows,sorted_cols,sorted_vals);

	delete[] row_ind,col_ind,values;
	row_ind = nullptr;
	col_ind = nullptr;
	values = nullptr;

	row_offset = new int[no_rows+1];
	int *counter = new int[no_rows];
	for(int i=0;i<no_rows;i++)
	{
		counter[i] = 0;
	}

	for(int i=0;i<nnz;i++)
	{
		counter[sorted_rows[i]] += 1;
	}

	row_offset[0] = 0;
	row_offset[no_rows] = nnz;

	for(int i = 1;i<=no_rows-1;i++)
	{
		row_offset[i] = row_offset[i-1] + counter[i-1];
	}
	
	delete[] counter;
	counter = nullptr;
	delete[] sorted_rows;
	sorted_rows = nullptr;
	
	x_glob = new double[no_cols];
	for(int i=0;i<no_cols;i++) x_glob[i] = i+1;

	int nb;
	int nt = 256;
	if(no_rows/nt < 1) nb = 1;
	else nb = ceil((double)no_rows/nt);
		
	double *d_result,*d_mat_vals,*d_vec_vals;
	int *d_row_offset,*d_col_inds;
	double *h_result = new double[no_rows];
	
	hipEvent_t start,stop;
	float milliseconds;

	hipMalloc(&d_result,no_rows*sizeof(double));
	hipMalloc(&d_mat_vals,nnz*sizeof(double));
	hipMalloc(&d_vec_vals,no_cols*sizeof(double));
	hipMalloc(&d_row_offset,(no_rows+1)*sizeof(int));
	hipMalloc(&d_col_inds,nnz*sizeof(int));

	double total_time = 0;

	for(int counter =0; counter < num_times_to_run; counter ++)
	{
		hipMemcpy(d_mat_vals,sorted_vals,nnz*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(d_vec_vals,x_glob,no_cols*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(d_row_offset,row_offset,(no_rows+1)*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_col_inds,sorted_cols,nnz*sizeof(int),hipMemcpyHostToDevice);

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);
		calc<<<nb,nt>>>(d_result,d_mat_vals,d_vec_vals,d_row_offset,d_col_inds,no_rows);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		total_time += milliseconds;

		hipMemcpy(h_result,d_result,no_rows*sizeof(double),hipMemcpyDeviceToHost);
	}
	printf("Average time = %lf microseconds\n",(total_time*1e3)/num_times_to_run);
	ofstream out_file_result;
	out_file_result.open("result.txt");
	for(int i=0;i<no_rows;i++)
	{
		out_file_result << h_result[i] << endl;
	}
	out_file_result.close();

	return 0;
}
